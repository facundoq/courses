#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>

/* Red Eye Removal
 ===============

 For this assignment we are implementing red eye removal.  This is
 accomplished by first creating a score for every pixel that tells us how
 likely it is to be a red eye pixel.  We have already done this for you - you
 are receiving the scores and need to sort them in ascending order so that we
 know which pixels to alter to remove the red eye.

 Note: ascending order == smallest to largest

 Each score is associated with a position, when you sort the scores, you must
 also move the positions accordingly.

 Implementing Parallel Radix Sort with CUDA
 ==========================================

 The basic idea is to construct a histogram on each pass of how many of each
 "digit" there are.   Then we scan this histogram so that we know where to put
 the output of each digit.  For example, the first 1 must come after all the
 0s so we have to know how many 0s there are to be able to start moving 1s
 into the correct position.

 1) Histogram of the number of occurrences of each digit
 2) Exclusive Prefix Sum of Histogram
 3) Determine relative offset of each digit
 For example [0 0 1 1 0 0 1]
 ->  [0 1 0 1 2 3 2]
 4) Combine the resluts of steps 2 & 3 to determine the final
 output location for each element and move it there

 LSB Radix sort is an out-of-place sort and you will need to ping-pong values
 between the input and output buffers we have provided.  Make sure the final
 sorted resluts end up in the output buffer!  Hint: You may need to do a copy
 at the end.

 */

/*
 *
 * Each block computes a different histogram. Histograms are merged afterwards.
 * Each block gets a portion of the input elements
 */
typedef unsigned long long int luint ;
__global__
void count_match_bitmask_kernel(unsigned int* const d_a, const size_t n, unsigned int bit_mask,
        luint* const d_count) {
    __shared__ luint block_count[1];

    if (threadIdx.x == 0) {
        block_count[0] = 0;
    }
    __syncthreads();

    luint total_threads = blockDim.x * gridDim.x;
    luint elements_per_thread = n / total_threads + ((n % total_threads) > 0);
    luint global_idx = (luint ) blockDim.x * (luint ) blockIdx.x
            + (luint) threadIdx.x;
    luint start = global_idx * elements_per_thread;

    if (start >= n) {
        //printf("t,b=%d,%d no work\n",blockIdx.x, threadIdx.x);
        return;
    }

    luint end = min((luint) n, start + elements_per_thread);
    luint count = 0;
    for (; start < end; start++) {
        if (d_a[start] & bit_mask) {
            count++;
        }
    }
    __syncthreads();
    atomicAdd(block_count, count);


    __syncthreads();
    if (threadIdx.x == 0) {
        //printf("block=%u,thread=%u,block_count=%llu,count=%llu, start=%llu, end=%llu, global_idx=%llu,el_per_thread=%llu,d_count=%llu\n",
          //      blockIdx.x, threadIdx.x, block_count[0], count, global_idx * elements_per_thread, end, global_idx,
            //    elements_per_thread,d_count[0]);
        atomicAdd(d_count, block_count[0]);
    }
}

const size_t count_max_threads_per_block = 1024;
const size_t max_blocks = 16;

/*
 * Computes the binary histogram of the entries in
 */
void count_match_bitmask(unsigned int* const d_a, const size_t n, luint* d_count, unsigned int bit_mask) {
    checkCudaErrors(hipMemset(d_count, 0, sizeof(luint) * 1));


    int threads = min(n, count_max_threads_per_block);
    size_t blocks_required_with_one_element_per_thread = n / threads + ((n % threads) > 0);
    int blocks = min(blocks_required_with_one_element_per_thread, max_blocks);
    const dim3 blockSize(threads);
    const dim3 gridSize(blocks);

    count_match_bitmask_kernel<<<gridSize, blockSize, 1>>>(d_a, n, bit_mask, d_count);
}

__global__
void copy_to_output_kernel(unsigned int* const  d_inputVals,unsigned int* const  d_outputPos,unsigned int* const d_outputVals,size_t n){
    luint total_threads = blockDim.x * gridDim.x;
    luint elements_per_thread = n / total_threads + ((n % total_threads) > 0);
    luint global_idx = (luint ) blockDim.x * (luint ) blockIdx.x
            + (luint) threadIdx.x;
    luint start = global_idx * elements_per_thread;

    if (start >= n) {
        //printf("t,b=%d,%d no work\n",blockIdx.x, threadIdx.x);
        return;
    }
    luint end = min((luint) n, start + elements_per_thread);

    for (; start < end; start++) {
        d_outputVals[d_outputPos[start]]=d_inputVals[start];
    }

}
void copy_to_output(unsigned int* const  d_inputVals,unsigned int* const  d_outputPos,unsigned int* const d_outputVals,size_t n){
    int threads = min(n, count_max_threads_per_block);
    size_t blocks_required_with_one_element_per_thread = n / threads + ((n % threads) > 0);
    int blocks = min(blocks_required_with_one_element_per_thread, max_blocks);
    const dim3 blockSize(threads);
    const dim3 gridSize(blocks);

    copy_to_output_kernel<<<gridSize, blockSize, 1>>>(d_inputVals,d_outputPos,d_outputVals,n);

}
__global__
void generate_output_positions_kernel(unsigned int* const  d_inputVals,unsigned int* const  d_outputPos,size_t n,unsigned int bit_mask,luint* d_indices){

}

void generate_output_positions(unsigned int* const  d_inputVals,unsigned int* const  d_outputPos,size_t n,unsigned int bit_mask,luint h_start_ones){

    luint* d_indices;
    checkCudaErrors(hipMalloc(&d_indices, sizeof(luint) * 2));
    checkCudaErrors(hipMemset(d_indices, 0, sizeof(luint) * 1));
    checkCudaErrors(hipMemset(&d_indices[1], h_start_ones, sizeof(luint) * 1));


    generate_output_positions_kernel<<<1,1,1>>>(d_inputVals,d_outputPos,n,bit_mask,d_indices);
}

void your_sort(unsigned int* d_inputVals, unsigned int* d_inputPos, unsigned int* d_outputVals,
        unsigned int* d_outputPos, size_t n) {

    unsigned int bits = sizeof(unsigned int) * 8;
    luint* d_count;
    checkCudaErrors(hipMalloc(&d_count, sizeof(luint) * 1));

    unsigned int bit_mask = 1;

    for (unsigned int bit_index = 0; bit_index < bits; bit_index++) {
        //printf("Bit index %d\n", bit_index);
        count_match_bitmask(d_inputVals, n, d_count, bit_mask);
        hipDeviceSynchronize();
        luint  h_count;
        checkCudaErrors(hipMemcpy(&h_count, d_count, sizeof(luint) * 1, hipMemcpyDeviceToHost));
        luint h_start_ones=n-h_count;
//        printf("d_Count %llu out of %lu\n", h_count, n);
//        h_count = 0;
//        unsigned int* h_inputVals = new unsigned int[n];
//        checkCudaErrors(hipMemcpy(h_inputVals, d_inputVals, sizeof(unsigned int) * n, hipMemcpyDeviceToHost));
//        for (size_t i = 0; i < n; i++) {
//            if (h_inputVals[i] & bit_mask) {
//                h_count++;
//            }
//        }
//        printf("h_count %llu out of %lu\n", h_count, n);

        //generate_output_positions(d_inputVals,d_outputPos,n,bit_mask,h_start_ones);
        // I refuse to code yet another blelloch scan with a slightly different input and output

        unsigned int* h_inputVals= new unsigned int[n];
        unsigned int* h_outputPos= new unsigned int[n];

        checkCudaErrors(hipMemcpy(h_inputVals, d_inputVals, sizeof(unsigned int) * n, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_outputPos, d_outputPos, sizeof(unsigned int) * n, hipMemcpyDeviceToHost));
        //printf("%lu/%lu start\n",h_start_ones,n);
        size_t zeros=0;
        size_t ones=h_start_ones;
        for (size_t i=0;i<n;i++){
            if ( (h_inputVals[i] & bit_mask)){
                h_outputPos[i]=ones;
                ones++;
            }else{
                h_outputPos[i]=zeros;
                zeros++;
            }
        }
        checkCudaErrors(hipMemcpy(d_outputPos, h_outputPos, sizeof(unsigned int) * n, hipMemcpyHostToDevice));

        copy_to_output(d_inputVals,d_outputPos,d_outputVals,n);
        // swap input and output
        unsigned int* temp=d_outputVals;
        d_outputVals=d_inputVals;
        d_inputVals=temp;
        free(h_inputVals);
        free(h_outputPos);
        bit_mask <<= 1;
    }
    checkCudaErrors(hipMemcpy(d_outputPos, d_inputPos, sizeof(unsigned int) * n, hipMemcpyDeviceToDevice));

}
