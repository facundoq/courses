#include "hip/hip_runtime.h"
/* Udacity Homework 3
 HDR Tone-mapping

 Background HDR
 ==============

 A High Dynamic Range (HDR) image contains a wider variation of intensity
 and color than is allowed by the RGB format with 1 byte per channel that we
 have used in the previous assignment.

 To store this extra information we use single precision floating point for
 each channel.  This allows for an extremely wide range of intensity values.

 In the image for this assignment, the inside of church with light coming in
 through stained glass windows, the raw input floating point values for the
 channels range from 0 to 275.  But the mean is .41 and 98% of the values are
 less than 3!  This means that certain areas (the windows) are extremely bright
 compared to everywhere else.  If we linearly map this [0-275] range into the
 [0-255] range that we have been using then most values will be mapped to zero!
 The only thing we will be able to see are the very brightest areas - the
 windows - everything else will appear pitch black.

 The problem is that although we have cameras capable of recording the wide
 range of intensity that exists in the real world our monitors are not capable
 of displaying them.  Our eyes are also quite capable of observing a much wider
 range of intensities than our image formats / monitors are capable of
 displaying.

 Tone-mapping is a process that transforms the intensities in the image so that
 the brightest values aren't nearly so far away from the mean.  That way when
 we transform the values into [0-255] we can actually see the entire image.
 There are many ways to perform this process and it is as much an art as a
 science - there is no single "right" answer.  In this homework we will
 implement one possible technique.

 Background Chrominance-Luminance
 ================================

 The RGB space that we have been using to represent images can be thought of as
 one possible set of axes spanning a three dimensional space of color.  We
 sometimes choose other axes to represent this space because they make certain
 operations more convenient.

 Another possible way of representing a color image is to separate the color
 information (chromaticity) from the brightness information.  There are
 multiple different methods for doing this - a common one during the analog
 television days was known as Chrominance-Luminance or YUV.

 We choose to represent the image in this way so that we can remap only the
 intensity channel and then recombine the new intensity values with the color
 information to form the final image.

 Old TV signals used to be transmitted in this way so that black & white
 televisions could display the luminance channel while color televisions would
 display all three of the channels.


 Tone-mapping
 ============

 In this assignment we are going to transform the luminance channel (actually
 the log of the luminance, but this is unimportant for the parts of the
 algorithm that you will be implementing) by compressing its range to [0, 1].
 To do this we need the cumulative distribution of the luminance values.

 Example
 -------

 input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
 min / max / range: 0 / 9 / 9

 histo with 3 bins: [4 7 3]

 cdf : [4 11 14]


 Your task is to calculate this cumulative distribution by following these
 steps.

 */

#include "utils.h"
#include <stdio.h>

/*
 a has length n
 reduce is computed in place per blocks
 so that a[blockIdx.x * blockDim.x] contains the minimum element of the block with indices
 (blockIdx.x * blockDim.x,blockIdx.x * blockDim.x)
 of the vector a
 */
__global__
void reduce_min_kernel(float* a, int n, bool isMin) {

    int block_offset = blockIdx.x * blockDim.x;
    int id = threadIdx.x;
    int offset = id + block_offset;
    // divide input in blocks: 0...elements_per_block-1,,elements_per_block..elements_per_block*2-1, elements_per_block*k..n
    int elements_in_extra_block = n % blockDim.x;
    int active_blocks = n / blockDim.x + (elements_in_extra_block != 0);
    int block_n; // elements per block
    if (blockIdx.x == active_blocks && elements_in_extra_block != 0) {
        block_n = elements_in_extra_block; // the last block does less work if n is not divisible by number of threads in block
    } else {
        block_n = blockDim.x;
    }
    int h = block_n / 2;

    while (id < h) {
//		if (id == 0) {
//			float m = a[0];
//			for (int i = 1; i < n; i++) {
//				m = max(m, a[i]);
//			}
//			printf("max=%d", m);
//		}
        //__syncthreads();

        // comparte two elements and put the min/max i
        if (isMin) {
            a[offset] = min(a[offset + h], a[offset]);
        } else {
            a[offset] = max(a[offset + h], a[offset]);
        }
        // if the block_current n is not divisible by 2, the first thread of the block updates with the last element
        if ((id == 0) && (block_n % 2 == 1)) {
            if (isMin) {
                a[block_offset] = min(a[block_offset], a[block_offset + block_n - 1]);
            } else {
                a[block_offset] = max(a[block_offset], a[block_offset + block_n - 1]);
            }
        }
        block_n = h;
        h /= 2;
        __syncthreads();
    }

}

__global__
void reduce_min_kernel_blocks(float* a, int n, bool isMin) {

    int elements_in_extra_block = n % blockDim.x;
    int active_blocks = n / blockDim.x + (elements_in_extra_block != 0);

    for (int i = 1; i < active_blocks; i++) {
        if (isMin) {
            a[0] = min(a[i * blockDim.x], a[0]);
        } else {
            a[0] = max(a[i * blockDim.x], a[0]);
        }

    }
}

const int elements_per_thread = 1;

// assumes n<= maximum_block_size*maximum_block_number
float reduce_min(const float* const d_a, int n, bool isMin) {
    const int BS = 1024;
    const dim3 blockSize(BS, 1, 1);

    const int elements_per_block = elements_per_thread * BS;
    int blocks = n / elements_per_block / 2;
    if (blocks * elements_per_block * 2 < n) {
        //printf("increase %d \n",blocks * elements_per_block );
        blocks++;
    }
    //printf("%d elements, %d blocks, %d elements_per_block\n", n, blocks,elements_per_block);

    const dim3 gridSize(blocks, 1, 1);
    float * d_min;
    hipDeviceSynchronize();
    checkCudaErrors(hipMalloc(&d_min, sizeof(float) * n));
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(d_min, d_a, sizeof(float) * n, hipMemcpyDeviceToDevice));
    reduce_min_kernel<<<gridSize, blockSize>>>(d_min, n, isMin);

    reduce_min_kernel_blocks<<<1, 1>>>(d_min, n, isMin);

    float result;
    checkCudaErrors(hipMemcpy(&result, d_min, sizeof(float), hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_min));
    return result;
}

__global__
void histogram_kernel(unsigned int* d_hist, int bins, const float* d_logLuminance, int n, float min_logLum,
        float max_logLum, float lumRange) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    int bin = (d_logLuminance[i] - min_logLum) / lumRange * bins;
    atomicAdd(&(d_hist[bin]), 1);
}

// assumes n<= maximum_block_size*maximum_block_number
void histogram(unsigned int* d_hist, int bins, const float* d_logLuminance, int n, float min_logLum, float max_logLum,
        float lumRange) {
    const int BS = 1024;
    const dim3 block_size(BS, 1, 1);
    const int blocks = (int) n / BS + (n % BS != 0);
    const dim3 grid_size(blocks, 1, 1);

    histogram_kernel<<<grid_size,block_size>>>(d_hist, bins, d_logLuminance, n, min_logLum, max_logLum, lumRange);
}
__global__
void blelloch_scan_sum_inplace_kernel_phase1(unsigned int* a, int n) {
    // 01 23 45 67
    //  1  3  5  7
    //     3     7
    //           7

    // index to update
    int update = threadIdx.x * 2 + 1; // because we are spawning n/2 threads

    // index to look for
    int displacement = 1;
    int look_for = update - displacement;
    int iterations = update;
    while (iterations % 2 == 1) {
//        printf("%d <- %d || %u <- %u \n",update,look_for,a[update],a[look_for]);
        a[update] += a[look_for];
        displacement *= 2;
        look_for = update - displacement;
        iterations /= 2;
        __syncthreads();
    }
    if (update ==n-1){
        a[update]=0;
    }


}

__global__
void blelloch_scan_sum_inplace_kernel_phase2(unsigned int* a, int n) {




    //               7
    //       3       7
    //   1   3   5   7
    // 0 1 2 3 4 5 6 7

    // 7: 3, 5, 6
    // 5: _, _, 4
    // 3: _, 1, 2
    // 1: _, _, 0

    //                                    15
    //               7                    15
    //       3       7        11          15
    //   1   3   5   7   9    11    13    15
    // 0 1 2 3 4 5 6 7 8 9 10 11 12 13 14 15

    //     n (size of sub arrays)
    //     16  8   4   2
    //     displacements
    //     8   4   2   1
    //     look fors (! or _ means not used)
    //15:  7  11  13  14
    //13:  5!  9! 11! 12
    //11:  3!  7!  9  10
    // 9:  1!  5!  7!  8
    // 7:  _   3   5   6
    // 5:  _   _   _   4
    // 3:  _   _   1   2
    // 1:  _   _   _   0

    // index to update
    int update = threadIdx.x * 2 + 1; // because we are spawning n/2 threads


    int displacement = n/2;

    while (n>1) {
        if ( (update+1)% n == 0){
            // index to look for
            int look_for = update - displacement;
//            printf("%d <- %d || %u <- %u \n",update,look_for,a[update],a[look_for]);

            // downsweep op
            int t =a[look_for];
            a[look_for]=a[update];
            a[update] += t;


        }

        n=displacement;
        displacement /= 2;
        __syncthreads();
    }


}

void blelloch_scan_sum_inplace(unsigned int* d_a, int n) {

    const int BS = 1024;
    //n=32;
    const int threads_needed=n/2;

    if (threads_needed > BS ) {
        printf("\n  histogram length: %d\n", n);
        printf("ERROR: current version of scan can only be implemented in 1 block with %d threads!",BS);
        exit(0);
    }
    const dim3 block_size(threads_needed, 1, 1);
    const dim3 grid_size(1, 1, 1);


    blelloch_scan_sum_inplace_kernel_phase1<<< grid_size,block_size>>>(d_a, n);

    //hipDeviceSynchronize();

//    printf("middle\n");


//      unsigned int* h_a = new unsigned int[n];
//    checkCudaErrors(hipMemcpy(h_a , d_a, sizeof(unsigned int) * n, hipMemcpyDeviceToHost));
//    for (int i = 0; i < n; i++) {
//        printf("%d,", h_a[i]);
//    }
//    printf("\n");

    hipDeviceSynchronize();
    blelloch_scan_sum_inplace_kernel_phase2<<<grid_size,block_size>>>(d_a, n);


//    unsigned int* h_a = new unsigned int[n];
//    checkCudaErrors(hipMemcpy(h_a , d_a, sizeof(unsigned int) * n, hipMemcpyDeviceToHost));
//
//    for (int i = 0; i < n; i++) {
//        printf("%d,", h_a[i]);
//    }
//    printf("\n");

}


void your_histogram_and_prefixsum(const float* const d_logLuminance, unsigned int* const d_cdf, float &min_logLum,
        float &max_logLum, const size_t numRows, const size_t numCols, const size_t numBins) {
    //TODO
    /*Here are the steps you need to implement
     1) find the minimum and maximum value in the input logLuminance channel
     store in min_logLum and max_logLum
     2) subtract them to find the range
     3) generate a histogram of all the values in the logLuminance channel using
     the formula: bin = (lum[i] - lumMin) / lumRange * numBins
     4) Perform an exclusive scan (prefix sum) on the histogram to get
     the cumulative distribution of luminance values (this should go in the
     incoming d_cdf pointer which already has been allocated for you)       */

//	int *d_hist;
//	checkCudaErrors(hipMalloc(&d_hist, sizeof(int) * numBins));
    int n = numRows * numCols;
    min_logLum = reduce_min(d_logLuminance, n, true);
    max_logLum = reduce_min(d_logLuminance, n, false);

    float lumRange = max_logLum - min_logLum;
    histogram(d_cdf, numBins, d_logLuminance, n, min_logLum, max_logLum, lumRange);
    blelloch_scan_sum_inplace(d_cdf, numBins);

}
