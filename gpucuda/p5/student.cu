#include "hip/hip_runtime.h"
/* Udacity HW5
 Histogramming for Speed

 The goal of this assignment is compute a histogram
 as fast as possible.  We have simplified the problem as much as
 possible to allow you to focus solely on the histogramming algorithm.

 The input values that you need to histogram are already the exact
 bins that need to be updated.  This is unlike in HW3 where you needed
 to compute the range of the data and then do:
 bin = (val - valMin) / valRange to determine the bin.

 Here the bin is just:
 bin = val

 so the serial histogram calculation looks like:
 for (i = 0; i < numElems; ++i)
 histo[val[i]]++;

 That's it!  Your job is to make it run as fast as possible!

 The values are normally distributed - you may take
 advantage of this fact in your implementation.

 */

#include "utils.h"
#include <thrust/host_vector.h>

__global__
void yourHisto(const unsigned int* const d_vals, //INPUT
        unsigned int* const d_histo,      //OUPUT
        int numVals,const unsigned int numBins) {

    __shared__ unsigned int histogram[1];
    // Initialize shared block array with local histogram to 0
    if (threadIdx.x==0){
            histogram[0]=0;
    }
    __syncthreads();

    //const int elems_per_thread = numVals/blockDim.x;
    //const int start= elems_per_thread * threadIdx.x;
    //int end= start+elems_per_thread;
    //if (threadIdx.x==(blockDim.x-1)){
      //  end=numVals;
    //}

    // get id and check if it is a valid index
    int count=0;
    for (int i=threadIdx.x;i<numVals;i+=blockDim.x){
         if (d_vals[i]==blockIdx.x){
             count++;
         }
    }
    // each thread updates its local histogram
    atomicAdd(&histogram[0], count);

    __syncthreads();
    // Copy shared block array with local histogram to global histogram
    if (threadIdx.x==0){
         d_histo[blockIdx.x]=histogram[0];

    }
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
        unsigned int* const d_histo,      //OUTPUT
        const unsigned int numBins,  unsigned int numElems) {
    //numElems=1024;

    const int threads_per_block=128;
    const int blocks = numElems /threads_per_block + ((numElems % threads_per_block) > 0) ;

    //printf("blocks %d, threads per block %d, numbins %d, numelems %d\n",blocks,threads_per_block,numBins,numElems);
    yourHisto<<<numBins, threads_per_block>>>(d_vals, d_histo, numElems,numBins);

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}
